#include "hip/hip_runtime.h"
#pragma once

#include "ParallelLayer.cuh"
__global__ void matrix_vector_prod(float* matrixData, float* vectorData, float* outputData, int size){
    //Get the row and column
    int row = threadIdx.x + blockDim.x * blockIdx.x;

    if(row < size){
        float output = 0;
        for(int i = 0; i < size; i++){
            output += vectorData[i] * matrixData[i + (row * size)];
        }
        outputData[row] = output;
    }
    
}

ParallelLayer::ParallelLayer(){};

void ParallelLayer::loadMatrix(float* matrixData, size_t size){
    this->size = size;

    //Malloc for the matrix
    hipMalloc(&gpuMatrix, size * size * sizeof(float));
    
    //Memcpy the matrix to the gpu
    hipMemcpy(gpuMatrix, matrixData, size * size * sizeof(float), hipMemcpyHostToDevice);

    //Malloc for the vector
    hipMalloc(&gpuVector, size * sizeof(float));
    
    //Malloc for the vector
    hipMalloc(&gpuResultantVector, size * sizeof(float));
}

float* ParallelLayer::forwardMatrix(float* vectorData){
    //Memcpy the vector to the gpu
    hipMemcpy(gpuVector, vectorData, size * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = std::ceil(size/static_cast<float>(threads));

    dim3 THREADS{threads};
    dim3 BLOCKS{blocks};

    matrix_vector_prod<<<BLOCKS, THREADS>>>(gpuMatrix, gpuVector, gpuResultantVector, static_cast<int>(size));

    float* returns = new float[static_cast<int>(size)];

    hipMemcpy(returns, gpuResultantVector, size * sizeof(float), hipMemcpyDeviceToHost);

    return returns;
}