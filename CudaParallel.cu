#include "hip/hip_runtime.h"
#pragma once
#include "CudaParallel.cuh"
#include <iostream>
#include "hip/hip_runtime_api.h"

__global__ void dotProduct(float* vectorACuda, float* vectorBCuda, float* vectorCCuda, int size){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    while (i < size){
        vectorCCuda[i] = vectorACuda[i] * vectorBCuda[i];
        i += blockDim.x * 256;//This is thread/block
    }
}


__device__ int cudaPow(int val, int pow){
    int returns = 1;
    for(int i = 0; i < pow; i++){
        returns *= val;
    }

    return returns;
}

__global__ void consolodateVector(float* vectorCuda, int level, int size){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int space = cudaPow(2, level);
    int diff = cudaPow(2, level-1);
    if(i < size && (i + diff < size) && i % space == 0){
        vectorCuda[i] = vectorCuda[i] + vectorCuda[i + diff];
    }
}



CudaParallel::CudaParallel(size_t size, float* vectorData){
    m_array_size = size;

    //Malloc on the GPU for Vector A
    hipMalloc(&vectorConstantCuda, m_array_size * sizeof(float));
    //Memcpy the data to the GPU
    hipMemcpy(vectorConstantCuda, vectorData, m_array_size * sizeof(float), hipMemcpyHostToDevice);

    //Malloc on the GPU for Vector B
    hipMalloc(&vectorVariableCuda, m_array_size * sizeof(float));

    //Malloc on the GPU for Vector B
    hipMalloc(&vectorOutputCuda, m_array_size * sizeof(float));

    //Create the Host Storage for the resultant vector
    vectorOutput = new float[m_array_size];
}

// you must first call the hipGetDeviceProperties() function, then pass 
// the devProp structure returned to this function:
int CudaParallel::getCudaCores(){  
    int deviceID;
    hipDeviceProp_t props;

    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&props, deviceID);
        
    int CUDACores = _ConvertSMVer2Cores(props.major, props.minor) * props.multiProcessorCount;

    return CUDACores;
}

float CudaParallel::dotVectors(float* vectorData){

    //Memcpy the output vector
    hipMemcpy(vectorVariableCuda, vectorData, m_array_size * sizeof(float), hipMemcpyHostToDevice);

    //Do the dot products
    dotProduct<<<std::ceil(1024/256.0), 256>>>(vectorConstantCuda, vectorVariableCuda, vectorOutputCuda, reinterpret_cast<int>(m_array_size));

    int level = 1;

    while(std::pow(2, level) <= m_array_size){
        consolodateVector<<<std::ceil(m_array_size/256.0), 256>>>(vectorOutputCuda, level, m_array_size);
        level++;
    }

    consolodateVector<<<std::ceil(m_array_size/256.0), 256>>>(vectorOutputCuda, level, m_array_size);

    float* resultant = new float();

    hipMemcpy(resultant, vectorOutputCuda, sizeof(float), hipMemcpyDeviceToHost);

    return *resultant;
}

