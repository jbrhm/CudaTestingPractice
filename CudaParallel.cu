#include "hip/hip_runtime.h"
#pragma once
#include "CudaParallel.cuh"

__global__ void dotProduct(float* vectorACuda, float* vectorBCuda, float* vectorCCuda, int size){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < size){
        vectorCCuda[i] = vectorACuda[i] * vectorBCuda[i];
    }
}

// __global__ void CudaParallel::consolodateVector(float* vectorCuda, int level){
//     if()

//     int i = threadIdx.x + blockIdx.x * blockDim.x;
//     int space = cudaPow(2, level);
//     if(i < m_array_size){
        
//     }
// }

__device__ int cudaPow(int val, int pow){
    int returns = 1;
    for(int i = 0; i < pow; i++){
        returns *= val;
    }

    return returns;
}


CudaParallel::CudaParallel(size_t size){
    m_array_size = size;
}

float* CudaParallel::dotVectors(float* vectorA, float* vectorB){
    //Create the pointers on the GPU to the data
    float* vectorACuda;
    float* vectorBCuda;
    float* vectorCCuda;

    //Create the Host Storage for the resultant vector
    float* vectorC = new float[m_array_size];

    //Malloc on the GPU for Vector A
    hipMalloc(&vectorACuda, m_array_size * sizeof(float));

    //Malloc on the GPU for Vector B
    hipMalloc(&vectorBCuda, m_array_size * sizeof(float));

    //Malloc on the GPU for Vector B
    hipMalloc(&vectorCCuda, m_array_size * sizeof(float));

    //Copy vector A to the GPU
    hipMemcpy(vectorACuda, vectorA, m_array_size * sizeof(float), hipMemcpyHostToDevice);

    //Copy vector B to the GPU
    hipMemcpy(vectorBCuda, vectorB, m_array_size * sizeof(float), hipMemcpyHostToDevice);

    //Do the dot products
    dotProduct<<<std::ceil(m_array_size/256.0), 256>>>(vectorACuda, vectorBCuda, vectorCCuda, reinterpret_cast<int>(m_array_size));

    hipMemcpy(vectorC, vectorCCuda, m_array_size * sizeof(float), hipMemcpyDeviceToHost);

    return vectorC;
}

