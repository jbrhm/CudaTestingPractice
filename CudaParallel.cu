#include "hip/hip_runtime.h"
#pragma once
#include "CudaParallel.cuh"
#include <iostream>

__global__ void dotProduct(float* vectorACuda, float* vectorBCuda, float* vectorCCuda, int size){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < size){
        vectorCCuda[i] = vectorACuda[i] * vectorBCuda[i];
    }
}


__device__ int cudaPow(int val, int pow){
    int returns = 1;
    for(int i = 0; i < pow; i++){
        returns *= val;
    }

    return returns;
}

__global__ void consolodateVector(float* vectorCuda, int level, int size){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int space = cudaPow(2, level);
    int diff = cudaPow(2, level-1);
    if(i < size && (i + diff < size) && i % space == 0){
        vectorCuda[i] = vectorCuda[i] + vectorCuda[i + diff];
    }
}



CudaParallel::CudaParallel(size_t size){
    m_array_size = size;
}

float CudaParallel::dotVectors(float* vectorA, float* vectorB){
    //Create the pointers on the GPU to the data
    float* vectorACuda;
    float* vectorBCuda;
    float* vectorCCuda;

    //Create the Host Storage for the resultant vector
    float* vectorC = new float[m_array_size];

    //Malloc on the GPU for Vector A
    hipMalloc(&vectorACuda, m_array_size * sizeof(float));

    //Malloc on the GPU for Vector B
    hipMalloc(&vectorBCuda, m_array_size * sizeof(float));

    //Malloc on the GPU for Vector B
    hipMalloc(&vectorCCuda, m_array_size * sizeof(float));

    //Copy vector A to the GPU
    hipMemcpy(vectorACuda, vectorA, m_array_size * sizeof(float), hipMemcpyHostToDevice);

    //Copy vector B to the GPU
    hipMemcpy(vectorBCuda, vectorB, m_array_size * sizeof(float), hipMemcpyHostToDevice);

    //Do the dot products
    dotProduct<<<std::ceil(m_array_size/256.0), 256>>>(vectorACuda, vectorBCuda, vectorCCuda, reinterpret_cast<int>(m_array_size));

    int level = 1;

    while(std::pow(2, level) <= m_array_size){
        consolodateVector<<<std::ceil(m_array_size/256.0), 256>>>(vectorCCuda, level, m_array_size);
        level++;
    }

    consolodateVector<<<std::ceil(m_array_size/256.0), 256>>>(vectorCCuda, level, m_array_size);

    hipMemcpy(vectorC, vectorCCuda, sizeof(float), hipMemcpyDeviceToHost);

    return *vectorC;
}

